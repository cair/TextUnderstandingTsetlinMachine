#include "hip/hip_runtime.h"
/*

Copyright (c) 2019 Ole-Christoffer Granmo

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

This code implements the Tsetlin Machine from paper arXiv:1804.01508
https://arxiv.org/abs/1804.01508

*/

#include "TsetlinMachineKernels.cuh"
#include "TsetlinMachine.cuh"
#include "TsetlinMachineConfig.cuh"
#include "GPUConfig.cuh"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/**********************************/
/***** Constructor/Destructor *****/
/**********************************/

TsetlinMachine::TsetlinMachine()
{
  hipMallocManaged(&class_sum, sizeof(int));
  hipMallocManaged(&ta_state, CLAUSES*LA_CHUNKS*STATE_BITS*sizeof(int));
  hipMallocManaged(&clause_output, CLAUSES*sizeof(int));
  hipMallocManaged(&clause_feedback, CLAUSES*sizeof(int));
  hipMallocManaged(&all_exclude, CLAUSES*sizeof(int));
  //hipMallocManaged(&la_feedback, LA_CHUNKS*sizeof(int));

  initialize();
}

TsetlinMachine::~TsetlinMachine()
{
  hipFree(class_sum);
  hipFree(ta_state);
  hipFree(clause_output);
  hipFree(clause_feedback); 
  hipFree(all_exclude);
}

void TsetlinMachine::initialize()
{
  // Initialize Tsetlin Automata states
  for (int j = 0; j < CLAUSES; ++j) {
    for (int k = 0; k < LA_CHUNKS; ++k) {
      for (int b = 0; b < STATE_BITS-1; ++b) {
        int id = j*LA_CHUNKS*STATE_BITS + k*STATE_BITS + b;                   
        ta_state[id] = ~0;
      }

      int id = j*LA_CHUNKS*STATE_BITS + k*STATE_BITS + STATE_BITS-1;
      ta_state[id] = 0;
    }
  }
}

/****************************/
/***** Public Functions *****/
/****************************/

void TsetlinMachine::update(hiprandState *devStates, int *Xi, int target, float s)
{
  initialize_clause_output<<<GRID_SIZE,BLOCK_SIZE>>>(clause_output);
  hipDeviceSynchronize();

  calculate_clause_output<<<GRID_SIZE,BLOCK_SIZE>>>(ta_state, clause_output, Xi);
  hipDeviceSynchronize();

  int sum = 0;
  for (int clause = 0; clause < CLAUSES; ++clause) {
    int sign = 1 - 2 * (clause & 1);
    sum += sign * clause_output[clause];
  }

  if (sum > THRESHOLD) {
    sum = THRESHOLD;
  } else if (sum < -THRESHOLD) {
    sum = -THRESHOLD;
  }

  generate_clause_feedback<<<GRID_SIZE,BLOCK_SIZE>>>(devStates, clause_feedback, sum, target);
  hipDeviceSynchronize();

  type_i_feedback<<<GRID_SIZE,BLOCK_SIZE>>>(devStates, ta_state, clause_feedback, clause_output,  Xi, s);
  hipDeviceSynchronize();

  type_ii_feedback<<<GRID_SIZE,BLOCK_SIZE>>>(ta_state, clause_feedback, clause_output, Xi);
  hipDeviceSynchronize();
}

int TsetlinMachine::get_state(int id)
{
  return ta_state[id];
}

int TsetlinMachine::score(int *Xi)
{
  initialize_clause_output_predict<<<GRID_SIZE,BLOCK_SIZE>>>(clause_output, all_exclude);
  hipDeviceSynchronize();

  calculate_clause_output_predict<<<GRID_SIZE,BLOCK_SIZE>>>(ta_state, clause_output, all_exclude, Xi);
  hipDeviceSynchronize();

  update_with_all_exclude<<<GRID_SIZE,BLOCK_SIZE>>>(clause_output, all_exclude);
  hipDeviceSynchronize();

  int sum = 0;
  for (int clause = 0; clause < CLAUSES; ++clause) {
    int sign = 1 - 2 * (clause & 1);
    sum += sign * clause_output[clause];
  }

  if (sum > THRESHOLD) {
      sum = THRESHOLD;
  } else if (sum < -THRESHOLD) {
      sum = -THRESHOLD;
  }

  return sum;
}

